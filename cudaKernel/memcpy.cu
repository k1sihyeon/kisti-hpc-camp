
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void PrintArray(int tid, int *A) {
	printf("A[%d] = %d\n", tid, A[tid]);
	A[tid] = A[tid] + 1;
	if (tid == 0)
		printf("=========================\n");
}

__global__ void Print(int *A) {

	int tid = threadIdx.x;
	PrintArray(tid, A);
}

int main() {
	int *d1_A, *d2_A, *h_A, *h_B;
	int size = 5; int i;
	
	h_A = (int*)malloc(size * sizeof(int));
	h_B = (int*)malloc(size * sizeof(int));
	
	for(i = 0; i < size; i++)
		h_A[i] = i;

	//device mem alloc
	hipSetDevice(0);
	hipMalloc((int **)&d1_A, size * sizeof(int));

	hipSetDevice(1);
	hipMalloc((int **)&d2_A, size * sizeof(int));
	//==

	//data transfer host -> dev0
	hipSetDevice(0);
	hipMemcpy(d1_A, h_A, size * sizeof(int), hipMemcpyHostToDevice); //cudaMemcpyDefault
	Print<<<1, 5>>>(d1_A);
	hipDeviceSynchronize();

	//data transfer dev0 -> 1
	hipMemcpy(d2_A, d1_A, size * sizeof(int), hipMemcpyDeviceToDevice); //
	hipSetDevice(1);
	Print<<<1, 5>>>(d2_A);
	hipDeviceSynchronize();

	//data transfer dev1 -> host
	hipMemcpy(h_B, d2_A, size * sizeof(int), hipMemcpyDeviceToHost);
	for(i = 0; i < size; i++)
		printf("h_B[%d] = %d\n", i, h_B[i]);
	
	hipFree(d1_A);
	hipFree(d2_A);
	
	hipDeviceReset();
	
	return 0;
}
