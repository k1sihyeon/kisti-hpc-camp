
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void nestedHelloWorld(int const, int);

int main(int argc, char **argv) {

	int blocksize = 8;
	int igrid = 1;
	int size = igrid * blocksize;

	if (argc > 1) {
 		igrid = atoi(argv[1]);
		size = igrid * blocksize;
	}

	dim3 block(blocksize , 1);
	dim3 grid((size + block.x - 1) / block.x , 1);
	
	printf("%s Execution Configration : grid %d block %d\n", argv[0], grid.x, block.x);
	nestedHelloWorld<<<grid, block>>>(block.x, 0);
	hipDeviceReset();

	return 0;
}

__global__ void nestedHelloWorld(int const iSize, int iDepth) {

	int tid = threadIdx.x;
	printf("Recursion = %d: Hello World from thread %d block %d\n", iDepth, tid, blockIdx.x);
	
	if(iSize == 1) return;

	int nthreads = iSize / 2;
	
	if ((tid == 0) && (nthreads > 0)) {
		nestedHelloWorld<<<1, nthreads>>>(nthreads, ++iDepth);
		printf("------> nested execution depth: %d\n", iDepth);
	}

}

