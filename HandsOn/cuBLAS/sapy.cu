#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define N (10)

int main(void){
	float h_x[N], h_y[N];
	float *d_x, *d_y;
	hipMalloc((float**)&d_x, N*sizeof(float));
	hipMalloc((float**)&d_y, N*sizeof(float));
	for(int i=0;i<N;i++){
		h_x[i]=10.0f,	h_y[i]=1.0f;
	}
//	cudaMemcpy(d_x, h_x, N*sizeof(float), cudaMemcpyDefault);
//	cudaMemcpy(d_y, h_y, N*sizeof(float), cudaMemcpyDefault);

	hipblasSetVector(N, sizeof(float), h_x, 1, d_x,1);
	hipblasSetVector(N, sizeof(float), h_y, 1, d_y,1);

	hipblasStatus_t stat;
	hipblasHandle_t handle;

	hipblasCreate(&handle);
	const float alpha=2.0;
	stat = hipblasSaxpy(handle, N, &alpha, d_x, 1, d_y, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS){
		printf("Error =======\n");
	    hipblasDestroy(handle);
    	hipFree(d_x), hipFree(d_y);
		return EXIT_FAILURE;
	}
	hipMemcpy(h_y, d_y, N*sizeof(float), hipMemcpyDefault);

	for(int i=0;i<N;i++)
		printf("%.2f ",h_y[i]);
	printf("\n");

	hipblasDestroy(handle);
	hipFree(d_x), hipFree(d_y);	
	return EXIT_SUCCESS;	
}

