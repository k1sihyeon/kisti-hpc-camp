/* cg_driver.c */
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <omp.h>

void matrix_file_test(void);

void sp_mv(int n, int nnz, int *row, int *col, double *a, double *x, double *b);
//void cg(int n, int nnz, int row[nnz], int col[nnz], double a[nnz], double rhs[n], double x[n]);
void convert_sys_to_ge(int nnz, int ge_nnz, int *in_row, int *in_col, double *in_a,
                       int *out_row, int *out_col, double *out_a);

int main(void)
{
  double start, finish;
  start=omp_get_wtime();
  matrix_file_test();
  finish=omp_get_wtime();
  printf("Elased Time : %lf seconds\n",finish-start);
}



void matrix_file_test(void)
{
  int n,m, nnz;
  int i;
  FILE *file;
  char temp[128];
//  file=fopen("../e30r5000/e30r5000_sort.txt","rt");  // one-base indexing  
  file=fopen("../bcsstk38/GEMM.txt","rt");			// zero-base indexing ==> 64 line comment out
  fgets(temp,sizeof(temp),file);
  printf("%d : %s",1, temp);

  fscanf(file,"%d %d %d",&n,&m,&nnz);
  printf("N : %d,  M : %d, NNZ : %d\n",n,m,nnz);

  if(n != m) {
    printf("Matrix is not square matrix\n");
    return;
  }



  int *row, *col;
  double *a, *x, *b;

  row=(int*)malloc(sizeof(int)*nnz);
  col=(int*)malloc(sizeof(int)*nnz);

  a=(double*)malloc(sizeof(double)*nnz);
  x=(double*)malloc(sizeof(double)*n);
  b=(double*)malloc(sizeof(double)*n);

  for(i=0;i<nnz;++i){
    fscanf(file,"%d %d %lg",&row[i],&col[i],&a[i]);
//    row[i] -= 1, col[i] -= 1;
  }
  fclose(file);

  // determine b vector
  for(i=0;i<n;++i)
    x[i]=1.0;
  sp_mv(n,nnz,row,col,a,x,b);


  // initial guess
  for(i=0;i<n;++i)
    x[i]=0.0;



  // step 1 : declare vars.
  hipsolverSpHandle_t cusolverH=NULL;
  hipsparseMatDescr_t descrA=NULL;
  hipsparseStatus_t cusparse_status = HIPSPARSE_STATUS_SUCCESS;
  hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;

  int *d_csrRow, *d_csrCol;
  double *d_csrA, *d_b, *d_x;
  int reorder=0, singularity=0;
  const double tol=1e-8;

  hipError_t cudaStat1=hipSuccess;
  hipError_t cudaStat2=hipSuccess;
  hipError_t cudaStat3=hipSuccess;
  hipError_t cudaStat4=hipSuccess;
  hipError_t cudaStat5=hipSuccess;

  // step 2 : create cusolver handle
  cusolver_status=hipsolverSpCreate(&cusolverH);
  assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

  cusparse_status = hipsparseCreateMatDescr(&descrA);
  assert(cusparse_status == HIPSPARSE_STATUS_SUCCESS);

  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);


  // step 3: allocate memory & copy data
  int *d_cooRowInd;
  cudaStat1 = hipMalloc((int**)&d_cooRowInd, sizeof(int)*nnz);
  cudaStat2 = hipMemcpy(d_cooRowInd, row, sizeof(int)*nnz, hipMemcpyHostToDevice);//Default);
  assert(cudaStat1==hipSuccess);
  assert(cudaStat2==hipSuccess);

  cudaStat1 = hipMalloc((int**)&d_csrRow, sizeof(int)*(n+1));
  cudaStat2 = hipMalloc((int**)&d_csrCol, sizeof(int)*nnz);
  cudaStat3 = hipMalloc((double**)&d_csrA, sizeof(double)*nnz);
  cudaStat4 = hipMalloc((double**)&d_b, sizeof(double)*n);
  cudaStat5 = hipMalloc((double**)&d_x, sizeof(double)*n);
  assert(cudaStat1==hipSuccess);
  assert(cudaStat2==hipSuccess);
  assert(cudaStat3==hipSuccess);
  assert(cudaStat4==hipSuccess);
  assert(cudaStat5==hipSuccess);
 
  // convert coo => csr 
  hipsparseHandle_t cusp_H;
  hipsparseCreate(&cusp_H);
  hipsparseXcoo2csr(cusp_H, d_cooRowInd, nnz, n, d_csrRow, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseDestroy(cusp_H);


   
  cudaStat1=hipMemcpy(d_csrCol, col, sizeof(int)*nnz, hipMemcpyHostToDevice);//Default);
  cudaStat2=hipMemcpy(d_csrA, a, sizeof(double)*nnz, hipMemcpyHostToDevice);//Default);
  cudaStat3=hipMemcpy(d_b, b, sizeof(double)*n, hipMemcpyHostToDevice);//Default);
  assert(cudaStat1==hipSuccess);
  assert(cudaStat2==hipSuccess);
  assert(cudaStat3==hipSuccess);

  cusolver_status = hipsolverSpDcsrlsvqr(cusolverH, n, nnz, descrA,
									d_csrA, d_csrRow, d_csrCol, d_b, tol, reorder, d_x, &singularity);


  assert(cusolver_status==HIPSOLVER_STATUS_SUCCESS);
  cudaStat4=hipMemcpy(x, d_x, sizeof(double)*n, hipMemcpyDefault);
  printf("%s\n",hipGetErrorName(cudaStat4));
  assert(cudaStat4 == hipSuccess);


  file=fopen("result_x.txt","wt");
  fprintf(file,"%s\n", "# i X ");

  for(i=0;i<n;++i){
    fprintf(file,"%d %lf\n",i,x[i]);
  }
  fclose(file);

  hipsparseDestroyMatDescr(descrA);
  hipsolverSpDestroy(cusolverH);

  hipFree(d_csrRow), hipFree(d_csrCol), hipFree(d_csrA), hipFree(d_b), hipFree(d_x);
  free(row), free(col), free(a), free(x), free(b);

}
