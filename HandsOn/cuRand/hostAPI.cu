/*
 * This program uses the host CURAND API to generate 100 
 * pseudorandom floats.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
int main(void)
{
	size_t n = 100;
	size_t i;
	hiprandGenerator_t gen;
	float *devData, *hostData;
	
	/* Allocate n floats on host */
	hostData=(float*)malloc(n*sizeof(float));
	hipMalloc((float**)&devData,n*sizeof(float));

	/* Create pseudo-random number generator */
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	/* Set seed */
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
	/* Generate n floats on device */
	hiprandGenerateUniform(gen, devData, n);
	
	/* Copy device memory to host */
	hipMemcpy(hostData, devData, n*sizeof(float), hipMemcpyDefault);

	/* Show result */
	for(i=0;i<n;++i)
		printf("%1.4f ", hostData[i]);
	printf("\n");

	/* Cleanup */
	hiprandDestroyGenerator(gen);
	hipFree(devData);
	free(hostData);

	return 0;
}
