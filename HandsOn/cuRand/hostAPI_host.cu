/*
 * This program uses the host CURAND API to generate 100 
 * pseudorandom floats.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
int main(void)
{
	size_t n = 100;
	size_t i;
	hiprandGenerator_t gen;
	float *hostData;
	
	/* Allocate n floats on host */
	hostData=(float*)malloc(n*sizeof(float));

	/* Create pseudo-random number generator */
	hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	/* Set seed */
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
	/* Generate n floats on device */
	hiprandGenerateUniform(gen, hostData, n);
	

	/* Show result */
	for(i=0;i<n;++i)
		printf("%1.4f ", hostData[i]);
	printf("\n");

	/* Cleanup */
	hiprandDestroyGenerator(gen);
	free(hostData);

	return 0;
}
