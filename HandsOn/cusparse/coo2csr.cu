#include <stdio.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <memory.h>
int main(void)
{

	hipsparseHandle_t handle;
	int n=5, nnz=13;
	int *d_cooRowInd, *d_csrRow;
    int cooRowInd[nnz], csrRow[n+1];
	hipMalloc((int**)&d_cooRowInd, sizeof(int)*nnz);
	hipMalloc((int**)&d_csrRow, sizeof(int)*(n+1));

	cooRowInd[0]=0,	cooRowInd[1]=0,	cooRowInd[2]=0,	cooRowInd[3]=0,	cooRowInd[4]=0;
	cooRowInd[5]=1,	cooRowInd[6]=1,	cooRowInd[7]=2,	cooRowInd[8]=2,	cooRowInd[9]=3;
	cooRowInd[10]=3,	cooRowInd[11]=4,	cooRowInd[12]=4;

	hipMemcpy(d_cooRowInd, cooRowInd, sizeof(int)*(nnz), hipMemcpyHostToDevice);
// 1. coo -> csr
	hipsparseCreate(&handle);
	hipsparseXcoo2csr(handle, d_cooRowInd, nnz,n,d_csrRow, HIPSPARSE_INDEX_BASE_ZERO);
	hipMemcpy(csrRow, d_csrRow,sizeof(int)*(n+1), hipMemcpyDeviceToHost);
	for(int i=0;i<n+1;++i)
		printf("%d ",csrRow[i]);

	printf("\n");

// 2. csr -> coo
	hipMemset(d_cooRowInd, 0, sizeof(int)*(nnz));
	memset(cooRowInd,0, sizeof(int)*(nnz));	
	hipsparseXcsr2coo(handle, d_csrRow, nnz, n, d_cooRowInd, HIPSPARSE_INDEX_BASE_ZERO);
	hipMemcpy(cooRowInd, d_cooRowInd, sizeof(int)*(nnz), hipMemcpyDeviceToHost);

	for(int i=0;i<nnz;++i)
		printf("%d ",cooRowInd[i]);
	printf("\n");		

    hipsparseDestroy(handle);
	hipFree(d_cooRowInd), hipFree(d_csrRow);
	return 0;
}
