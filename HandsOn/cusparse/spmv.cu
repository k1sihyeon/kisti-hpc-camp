#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
int main(void)
{
	hipError_t cudaStat=hipSuccess;
	const int n=5, nnz=13;
	int *d_cooRowInd, *d_cooColInd;
	float *d_A;
	int cooRowInd[nnz]={0,0,0,0,0,1,1,2,2,3,3,4,4};
	int cooColInd[nnz]={0,1,2,3,4, 0,1, 0,2, 0,3, 0,4 };
	float A[nnz]={4.0f, 1.0f, 2.0f, 0.5f, 2.0f, 1.0f, 0.5f, 2.0f, 3.0f, 0.5f, 0.625f, 2.0f, 16.0f};

	cudaStat=hipMalloc((int**)&d_cooRowInd, nnz*sizeof(int));
	if(cudaStat != hipSuccess) printf("bbb\n");
	cudaStat=hipMalloc((int**)&d_cooColInd, nnz*sizeof(int));

	if(cudaStat != hipSuccess){
		printf("aaaa\n");

	}
	
	hipMalloc((float**)&d_A, nnz*sizeof(float));
	hipMemcpy(d_cooRowInd, cooRowInd, nnz*sizeof(int), hipMemcpyDefault);
	hipMemcpy(d_cooColInd, cooColInd, nnz*sizeof(int), hipMemcpyDefault);
	hipMemcpy(d_A, A, nnz*sizeof(float), hipMemcpyDefault);
	float alpah=1.0f, beta=1.0f;


// Vector initialize
	float x[n]={1.0f, 1.0f, 1.0f, 1.0f, 1.0f};
	float y[n]={1.0f, 1.0f, 1.0f, 1.0f, 1.0f};
	float hY[n];
	float *d_x, *d_y;
	hipMalloc((float**)&d_x, sizeof(float)*n);
	hipMalloc((float**)&d_y, sizeof(float)*n);
	hipMemcpy(d_x, x, sizeof(float)*n, hipMemcpyDefault);
	hipMemcpy(d_y, y, sizeof(float)*n, hipMemcpyDefault);

    hipsparseHandle_t handle=NULL;
    hipsparseStatus_t stat;
	size_t bufferSize=0;
	hipsparseCreate(&handle);
	hipsparseSpMatDescr_t matA;
	hipsparseDnVecDescr_t vecX, vecY;

	hipsparseCreateCoo(&matA, n,n,nnz,d_cooRowInd, d_cooColInd,d_A, HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_32F);

	hipsparseCreateDnVec(&vecX, n, d_x, HIP_R_32F);
	hipsparseCreateDnVec(&vecY, n, d_y, HIP_R_32F);

	hipsparseSpMV_bufferSize(	handle,
								HIPSPARSE_OPERATION_NON_TRANSPOSE,
								&alpah,
								matA,
								vecX,
								&beta,
								vecY,
								HIP_R_32F,
								HIPSPARSE_MV_ALG_DEFAULT,
//								HIPSPARSE_SPMV_COO_ALG1,
								&bufferSize);

	printf("bufferSize: %zd\n",bufferSize);
	void *dBuffer;
	hipMalloc(&dBuffer, bufferSize);


	hipsparseSpMV(	handle,
					HIPSPARSE_OPERATION_NON_TRANSPOSE,
					(float*)&alpah,
					matA,
					vecX,
					(float*)&beta,
					vecY,
					HIP_R_32F,
					HIPSPARSE_MV_ALG_DEFAULT,
					dBuffer
				);

	hipMemcpy(hY,d_y,sizeof(float)*n, hipMemcpyDefault);
	for(int i=0;i<n;i++)
		printf("%f ",(float)hY[i]);
	printf("\n");

	hipsparseDestroyDnVec(vecX);
	hipsparseDestroyDnVec(vecY);
	hipsparseDestroySpMat(matA);
	hipsparseDestroy(handle);

	return EXIT_SUCCESS;
}
